#include "hip/hip_runtime.h"
/* 
* Copyright (c) 2018 NVIDIA CORPORATION. All rights reserved.
*
* Redistribution and use in source and binary forms, with or without
* modification, are permitted provided that the following conditions
* are met:
*  * Redistributions of source code must retain the above copyright
*    notice, this list of conditions and the following disclaimer.
*  * Redistributions in binary form must reproduce the above copyright
*    notice, this list of conditions and the following disclaimer in the
*    documentation and/or other materials provided with the distribution.
*  * Neither the name of NVIDIA CORPORATION nor the names of its
*    contributors may be used to endorse or promote products derived
*    from this software without specific prior written permission.
*
* THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
* EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
* IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
* PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
* CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
* EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
* PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
* PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
* OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
* (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
* OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
*/

#include <optix.h>
#include <optix_math.h>
#include <optixu/optixu_math_namespace.h>
#include <optixu/optixu_matrix_namespace.h>

static __device__ __inline__ float3 schlick( float nDi, const float3& rgb )
{
float r = fresnel_schlick(nDi, 5, rgb.x, 1);
float g = fresnel_schlick(nDi, 5, rgb.y, 1);
float b = fresnel_schlick(nDi, 5, rgb.z, 1);
return make_float3(r, g, b);
}

struct PerRayData_radiance
{
float3 result;
float importance;
int depth;
};

struct PerRayData_shadow
{
float3 attenuation;
};

struct BasicLight
{
#if defined(__cplusplus)
typedef optix::float3 float3;
#endif
float3 pos;
float3 color;
int    casts_shadow; 
int    padding;      // make this structure 32 bytes -- powers of two are your friend!
};

using namespace optix;

rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );
rtDeclareVariable(float, t_hit, rtIntersectionDistance, );
rtDeclareVariable(PerRayData_radiance, prd_radiance, rtPayload, );
rtDeclareVariable(PerRayData_shadow,   prd_shadow, rtPayload, );

rtDeclareVariable(float3,       Ka, , );
rtDeclareVariable(float3,       Ks, , );
rtDeclareVariable(float,        dissolve, , );
rtDeclareVariable(float,        phong_exp, , );
rtDeclareVariable(float3, 		reflectivity_n, , );

rtDeclareVariable(float3, geometric_normal, attribute geometric_normal, ); 
rtDeclareVariable(float3, shading_normal, attribute shading_normal, ); 
rtDeclareVariable(float3, shading_tangent, attribute shading_tangent, ); 
rtDeclareVariable(float3, shading_bitangent, attribute shading_bitangent, ); 

rtDeclareVariable(int,               max_depth, , );
rtBuffer<BasicLight>                 lights;
rtDeclareVariable(float3,            ambient_light_color, , );
rtDeclareVariable(unsigned int,      radiance_ray_type, , );
rtDeclareVariable(unsigned int,      shadow_ray_type, , );
rtDeclareVariable(float,             scene_epsilon, , );
rtDeclareVariable(rtObject,          top_object, , );
rtDeclareVariable(rtObject,          top_shadower, , );

RT_PROGRAM void any_hit_shadow()
{
    if(dissolve < 1.0)
    {
        prd_shadow.attenuation = optix::make_float3(1.0f);
        rtIgnoreIntersection();
    }
    else
    {
        // this material is opaque, so it fully attenuates all shadow rays
        prd_shadow.attenuation = optix::make_float3(0.0f);
        rtTerminateRay();
    }
}

rtTextureSampler<float4, 2> Kd_map;
rtTextureSampler<float4, 2> Kr_map;
rtTextureSampler<float4, 2> Kn_map;
rtDeclareVariable(float3, texcoord, attribute texcoord, ); 

RT_PROGRAM void closest_hit_radiance()
{
    float3 world_geometric_normal	= normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, geometric_normal ) );
    float3 world_shading_tangent	= normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, shading_tangent ) );
    float3 world_shading_bitangent	= normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, shading_bitangent ) );
    float3 world_shading_normal		= normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, shading_normal ) );

    float3 fftangent = faceforward( world_shading_tangent, -ray.direction, world_geometric_normal );
    float3 ffbitangent = faceforward( world_shading_bitangent, -ray.direction, world_geometric_normal );
    float3 ffnormal = faceforward( world_shading_normal, -ray.direction, world_geometric_normal );

    Matrix<3, 3> invtangentMatrix;
    invtangentMatrix.setCol(0, fftangent);
    invtangentMatrix.setCol(1, ffbitangent);
    invtangentMatrix.setCol(2, ffnormal);

    const float3 Kd_val = make_float3( tex2D( Kd_map, texcoord.x, texcoord.y ) );
    const float3 Kr_val = make_float3( tex2D( Kr_map, texcoord.x, texcoord.y ) );
    const float3 Kn_val = invtangentMatrix * normalize(make_float3( tex2D( Kn_map, texcoord.x, texcoord.y ) ) * 2.0f - 1.0f);

    float3 hit_point = ray.origin + t_hit * ray.direction;

    // ambient contribution

    float3 result = Ka * ambient_light_color;

    // reflection ray
    if(dissolve < 1.0) 
    {
        // ray tree attenuation
        PerRayData_radiance new_prd;             
        new_prd.importance = prd_radiance.importance;
        new_prd.depth = prd_radiance.depth + 1;

        optix::Ray glass_ray = optix::make_Ray( hit_point, ray.direction, radiance_ray_type, scene_epsilon, RT_DEFAULT_MAX );
        rtTrace(top_object, glass_ray, new_prd);
        result = (result * dissolve) + (new_prd.result * (1.0 - dissolve));
    }

    // compute direct lighting
    unsigned int num_lights = lights.size();
    for(int i = 0; i < num_lights; ++i) 
    {
        BasicLight light = lights[i];
        float Ldist = optix::length(light.pos - hit_point);
        float3 L = optix::normalize(light.pos - hit_point);
        float nDl = optix::dot( Kn_val, L);

        // cast shadow ray
        float3 light_attenuation = make_float3(static_cast<float>( nDl > 0.0f ));
        if ( nDl > 0.0f && light.casts_shadow ) 
        {
            PerRayData_shadow shadow_prd;
            shadow_prd.attenuation = make_float3(1.0f);
            optix::Ray shadow_ray = optix::make_Ray( hit_point, L, shadow_ray_type, scene_epsilon, Ldist );
            rtTrace(top_shadower, shadow_ray, shadow_prd);
            light_attenuation = shadow_prd.attenuation;
        }

        // If not completely shadowed, light the hit point
        if( fmaxf(light_attenuation) > 0.0f ) 
        {
            float3 Lc = light.color * light_attenuation;

            result += Kd_val * nDl * Lc;

            float3 H = optix::normalize(L - ray.direction);
            float nDh = optix::dot( Kn_val, H );
            if(nDh > 0) 
            {
                float power = pow(nDh, phong_exp);
                result += Kr_val * power * Lc;
            }
        }
    }

    // fresnel reflectance
    float3 r = schlick(-dot(Kn_val, ray.direction), reflectivity_n);

    // calculate reflection
    if( prd_radiance.importance >= 0.01f && prd_radiance.depth <= max_depth ) 
    {
        // ray tree attenuation
        PerRayData_radiance new_prd;             
        new_prd.importance = (prd_radiance.importance * optix::luminance( r ));
        new_prd.depth = prd_radiance.depth + 1;

        // reflection ray
        if( new_prd.importance >= 0.01f && new_prd.depth <= max_depth) 
        {
            float3 R = optix::reflect( ray.direction, Kn_val );
            optix::Ray refl_ray = optix::make_Ray( hit_point, R, radiance_ray_type, scene_epsilon, RT_DEFAULT_MAX );
            rtTrace(top_object, refl_ray, new_prd);
            result +=  (r * new_prd.result);
        }
    }
    
    // pass the color back up the tree
    prd_radiance.result = result;
}