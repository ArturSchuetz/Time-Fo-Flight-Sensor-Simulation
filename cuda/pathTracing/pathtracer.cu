#include "hip/hip_runtime.h"
/* 
* Copyright (c) 2018 NVIDIA CORPORATION. All rights reserved.
*
* Redistribution and use in source and binary forms, with or without
* modification, are permitted provided that the following conditions
* are met:
*  * Redistributions of source code must retain the above copyright
*    notice, this list of conditions and the following disclaimer.
*  * Redistributions in binary form must reproduce the above copyright
*    notice, this list of conditions and the following disclaimer in the
*    documentation and/or other materials provided with the distribution.
*  * Neither the name of NVIDIA CORPORATION nor the names of its
*    contributors may be used to endorse or promote products derived
*    from this software without specific prior written permission.
*
* THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
* EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
* IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
* PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
* CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
* EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
* PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
* PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
* OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
* (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
* OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
*/

#include <optix.h>
#include <optix_math.h>
#include <optixu/optixu_math_namespace.h>
#include <optixu/optixu_matrix_namespace.h>
#include "random.h"
#include "helpers.h"
#include "microfacet.h"

struct PerRayData_radiance
{
    float3 result;
    float3 radiance;
    float3 attenuation;

    float3 origin;
    float3 direction;

    float current_index_of_refraction;
    unsigned int seed;
    int depth;
    int done;
};

struct PerRayData_shadow
{
    bool inShadow;
};

struct BasicLight
{
    #if defined(__cplusplus)
    typedef optix::float3 float3;
    #endif
    float3 pos;
    float3 color;
    int    casts_shadow; 
    int    padding;      // make this structure 32 bytes -- powers of two are your friend!
};

using namespace optix;

// Scene wide variables

rtDeclareVariable(uint2, launch_index, rtLaunchIndex, );
rtDeclareVariable(uint2, launch_dim,   rtLaunchDim, );
rtDeclareVariable(float, time_view_scale, , ) = 1e-6f;


//-----------------------------------------------------------------------------
//  Camera program -- main ray tracing loop
//-----------------------------------------------------------------------------

rtDeclareVariable(float3,        eye, , );
rtDeclareVariable(float3,        U, , );
rtDeclareVariable(float3,        V, , );
rtDeclareVariable(float3,        W, , );
rtDeclareVariable(float3,        bad_color, , );
rtDeclareVariable(float3, 		 bg_color, , );
rtDeclareVariable(unsigned int,  frame_number, , );
rtDeclareVariable(unsigned int,  sqrt_num_samples, , );
rtDeclareVariable(unsigned int,  rr_begin_depth, , );

rtBuffer<float4, 2> input_rayDirections;
rtBuffer<float4, 2> output_buffer;

rtDeclareVariable(float3, back_hit_point,   attribute back_hit_point, ); 
rtDeclareVariable(float3, front_hit_point,  attribute front_hit_point, ); 
rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );

rtDeclareVariable(float, t_hit, rtIntersectionDistance, );
rtDeclareVariable(PerRayData_radiance, prd_radiance, rtPayload, );
rtDeclareVariable(PerRayData_shadow,   prd_shadow, rtPayload, );

rtDeclareVariable(float3, geometric_normal, attribute geometric_normal, ); 
rtDeclareVariable(float3, shading_normal, attribute shading_normal, ); 
rtDeclareVariable(float3, shading_tangent, attribute shading_tangent, ); 
rtDeclareVariable(float3, shading_bitangent, attribute shading_bitangent, ); 

rtBuffer<BasicLight>                 lights;
rtDeclareVariable(unsigned int,      radiance_ray_type, , );
rtDeclareVariable(unsigned int,      shadow_ray_type, , );
rtDeclareVariable(float,             scene_epsilon, , );
rtDeclareVariable(rtObject,          top_object, , );
rtDeclareVariable(rtObject,          top_shadower, , );

const float smallest_value = 0.01f; 

//-----------------------------------------------------------------------------
//
//  Camera program -- main ray tracing loop
//
//-----------------------------------------------------------------------------
/*
static __device__ __inline__ void random_concentric_sample_hemisphere(float u, float v, float3& p)
{
    float r = 0.0f;
    float phi = 0.0f;

    u = (2.0f * u) - 1.0f;
    v = (2.0f * v) - 1.0f;

    if (u > -v)
    {
        if (u > v)
        {
            r = u;
            phi = (M_PIf / 4.0f) * (v / u);
        }
        else
        {
            r = v;
            phi = (M_PIf / 4.0f) * (2.0f - (u / v));
        }
    }
    else
    {
        if(u < v)
        {
            r = -u;
            phi = (M_PIf / 4.0f) * (4.0f + (v / u));
        }
        else
        {
            r = -v;
            if (v != 0.0f)
            {                
                phi = (M_PIf / 4.0f) * (6.0f - (u / v));
            }
            else
            {
                phi = 0.0f;
            }
        }
    }
    // Project up to hemisphere.
    p.x = r * cosf( phi );
    p.y = r * sinf( phi );
    p.z = sqrtf( fmaxf( 0.0f, 1.0f - (p.x * p.x) - (p.y * p.y) ) );
}
*/

RT_PROGRAM void pathtrace_pinhole_camera()
{
    size_t2 screen = output_buffer.size();

    float2 inv_screen = 1.0f/make_float2(screen) * 2.f;
    float2 pixel = (make_float2(launch_index)) * inv_screen - 1.f;

    float2 jitter_scale = inv_screen / sqrt_num_samples;
    unsigned int samples_per_pixel = sqrt_num_samples*sqrt_num_samples;
    float3 result = make_float3(0.0f);

    size_t2 bufferSize = input_rayDirections.size();
    unsigned int seed = tea<16>(screen.x * launch_index.y + launch_index.x, frame_number);
    do 
    {
        //
        // Sample pixel
        //
        float2 jitter = make_float2(0.0f);
        do
        {
            jitter = make_float2((rnd(seed) * 2.0f) - 1.0f, (rnd(seed) * 2.0f) - 1.0f);
        } while(sqrt((jitter.x * jitter.x) + (jitter.y * jitter.y)) > 1.0f);

        float2 d = pixel + (jitter * inv_screen * 1.0f);

        if(d.x > 1.0f) d.x = 1.0f;
        if(d.y > 1.0f) d.y = 1.0f;
        if(d.x < -1.0f) d.x = -1.0f;
        if(d.y < -1.0f) d.y = -1.0f;

        //
        // Sample pixel using jittering
        //
        /*
        unsigned int x = samples_per_pixel % sqrt_num_samples;
        unsigned int y = samples_per_pixel / sqrt_num_samples;
        float2 jitter = make_float2(x - rnd(seed), y - rnd(seed));
        float2 d = pixel + jitter * jitter_scale;
        */

        float2 relativeCoordinate = ((d + 1.0f) / 2.0f);
        float3 calculated_ray = make_float3(input_rayDirections[make_uint2(relativeCoordinate.x * (bufferSize.x - 1), relativeCoordinate.y * (bufferSize.y - 1))]);

        float3 ray_origin = eye;
        float3 ray_direction = normalize(calculated_ray.x*U + calculated_ray.y*V + -calculated_ray.z*W);

        // Initialze per-ray data
        PerRayData_radiance prd;
        prd.result = make_float3(0.f);
        prd.attenuation = make_float3(1.f);

        prd.current_index_of_refraction = 1.00029f; // ior of air
        prd.seed = seed;
        prd.depth = 0;
        prd.done = false;

        // Each iteration is a segment of the ray path.  The closest hit will
        // return new segments to be traced here.
        for(;;)
        {
            Ray ray = make_Ray(ray_origin, ray_direction, radiance_ray_type, scene_epsilon, RT_DEFAULT_MAX);
            rtTrace(top_object, ray, prd);

            if(prd.done)
            {
                // We have hit the background or a luminaire
                prd.result += prd.radiance;
                break;
            }

            // Russian roulette termination 
            if(prd.depth >= rr_begin_depth)
            {
                float pcont = fmaxf(prd.attenuation);
                if(rnd(prd.seed) >= pcont)
                    break;
                    
                prd.attenuation /= pcont;
            }

            prd.depth++;
            prd.result += prd.radiance;

            // Update ray data for the next path segment
            ray_origin = prd.origin;
            ray_direction = prd.direction;
        }

        result += prd.result;

        seed = prd.seed;
    } while (--samples_per_pixel);

    //
    // Update the output buffer
    //
    float3 pixel_color = result/(sqrt_num_samples*sqrt_num_samples);

    if (frame_number > 1)
    {
        float a = 1.0f / (float)frame_number;
        float3 old_color = make_float3(output_buffer[launch_index]);
        output_buffer[launch_index] = make_float4( lerp( old_color, pixel_color, a ), 1.0f );
    }
    else
    {
        output_buffer[launch_index] = make_float4(pixel_color, 1.0f);
    }
}

rtTextureSampler<float4, 2> Kd_map;
rtTextureSampler<float4, 2> Ks_map;
rtTextureSampler<float4, 2> Kn_map;
rtTextureSampler<float4, 2> Tr_map;
rtTextureSampler<float4, 2> Pm_map;
rtTextureSampler<float4, 2> Ke_map;
rtDeclareVariable(float3, texcoord, attribute texcoord, ); 
rtDeclareVariable(float, index_of_refraction, , );
rtDeclareVariable(float, absorption_coefficien, , );

RT_PROGRAM void any_hit_shadow()
{
    const float3 Tr_val = make_float3( tex2D( Tr_map, texcoord.x, texcoord.y ) );
    if(Tr_val.x < 1.0f)
    {
        rtIgnoreIntersection();
    }
    else
    {
        prd_shadow.inShadow = true;
        rtTerminateRay();
    }
}

//-----------------------------------------------------------------------------
//
//  BRDF based on Cook–Torrance and Oren-Nayar
//
//-----------------------------------------------------------------------------

RT_PROGRAM void microfacet_closest_hit()
{
    float3 world_geometric_normal	= normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, geometric_normal ) );
    float3 world_shading_tangent	= normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, shading_tangent ) );
    float3 world_shading_bitangent	= normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, shading_bitangent ) );
    float3 world_shading_normal		= normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, shading_normal ) );

    float3 fftangent = faceforward( world_shading_tangent, -ray.direction, world_geometric_normal );
    float3 ffbitangent = faceforward( world_shading_bitangent, -ray.direction, world_geometric_normal );
    float3 ffnormal = faceforward( world_shading_normal, -ray.direction, world_geometric_normal );

    Matrix<3, 3> tangentMatrix;
    tangentMatrix.setCol(0, fftangent);
    tangentMatrix.setCol(1, ffbitangent);
    tangentMatrix.setCol(2, ffnormal);

    const float3 Kd_val = make_float3( tex2D( Kd_map, texcoord.x, texcoord.y ) );
    const float3 Ks_val = make_float3( tex2D( Ks_map, texcoord.x, texcoord.y ) );
          float3 Kn_val = normalize(make_float3( tex2D( Kn_map, texcoord.x, texcoord.y ) ) * 2.0f - 1.0f);
    const float3 Tr_val = make_float3( tex2D( Tr_map, texcoord.x, texcoord.y ) );
    const float3 Pm_val = make_float3( tex2D( Pm_map, texcoord.x, texcoord.y ) );
    const float3 Ke_val = make_float3( tex2D( Ke_map, texcoord.x, texcoord.y ) );

    Kn_val = tangentMatrix * make_float3(Kn_val.x, Kn_val.y, Kn_val.z);

    if(length(Kn_val) > 0)
        Kn_val = normalize(Kn_val);
    else
        Kn_val = ffnormal;

    if(Tr_val.x < 1.0f)
    {       
        if(prd_radiance.depth > 0 && Tr_val.x == 0.0f)
            prd_radiance.depth = prd_radiance.depth - 1;

        prd_radiance.attenuation = prd_radiance.attenuation * (1.0f - Tr_val.x);
        
        const float3 bhp = rtTransformPoint(RT_OBJECT_TO_WORLD, back_hit_point);
        prd_radiance.origin = bhp;
        prd_radiance.direction = ray.direction;

        prd_radiance.radiance = make_float3(0.0f);
        return;
    }
    
    float roughness = saturate(1.0f - Ks_val.x);
    float metallic = Pm_val.x;

    float3 hit_point = ray.origin + t_hit * ray.direction;


    //
    // Next event estimation (compute direct lighting).
    //
    float3 result = make_float3(0.0f);

    unsigned int num_lights = lights.size();
    for(int i = 0; i < num_lights; ++i)
    {
        // Choose random point on light
        BasicLight light = lights[i];

        // Calculate properties of light sample (for area based pdf)
        float3 lightDir = light.pos - hit_point;
        const float LightdistPow2 = dot(lightDir, lightDir);
        const float Lightdist = sqrt(LightdistPow2);
        lightDir = lightDir / Lightdist;

        const float NdotL = saturate(dot(Kn_val, lightDir));

        // cast shadow ray
        if ( NdotL > smallest_value)
        {
            PerRayData_shadow shadow_prd;
            shadow_prd.inShadow = false;
            // Note: bias both ends of the shadow ray, in case the light is also present as geometry in the scene.
            Ray shadow_ray = make_Ray( hit_point, lightDir, shadow_ray_type, scene_epsilon, Lightdist - scene_epsilon);
            rtTrace(top_object, shadow_ray, shadow_prd);

            if(!shadow_prd.inShadow)
            {
                float3 fresnel;
                if(metallic >= 0.99f)
                {
                    fresnel = Kd_val * FrConductor(dot(-ray.direction, lightDir), prd_radiance.current_index_of_refraction, index_of_refraction, absorption_coefficien);
                }
                else if(metallic <= 0.01f)
                {
                    fresnel = make_float3(FrDielectric(dot(-ray.direction, lightDir), prd_radiance.current_index_of_refraction, index_of_refraction));
                }
                else
                {
                    fresnel = make_float3(0.0f);
                }
        
                // =====================================
                // Visible light
                float3 diffuse = OrenNayar_full_f(Kd_val, Kn_val, -ray.direction, lightDir, roughness);
                diffuse = diffuse * (make_float3(1.0f) - fresnel) * (1.0f - metallic);

                float3 specular = TorranceSparrow_f(Kn_val, -ray.direction, lightDir, fresnel, roughness);
            
                result += prd_radiance.attenuation * (diffuse + specular) * (light.color * (NdotL / LightdistPow2));
            }
        }
    }

    prd_radiance.radiance = result;
    
    if(Ke_val.x > 0.0f || Ke_val.y > 0.0f || Ke_val.z > 0.0f)
    {
        prd_radiance.radiance += prd_radiance.attenuation * Ke_val;
    }

    //
    // Generate a reflection ray.  This will be traced back in ray-gen.
    //
    prd_radiance.origin = hit_point;

    float3 p;

    optix::Onb onb( Kn_val );
    float z1=rnd(prd_radiance.seed);
    float z2=rnd(prd_radiance.seed);
    random_sample_hemisphere(z1, z2, p);

    onb.inverse_transform(p);    
    prd_radiance.direction = normalize(p);
        
    const float NdotL = saturate(dot(Kn_val, prd_radiance.direction));
    if (NdotL > smallest_value)
    {
        // Calculate fresnel
        float3 fresnel;
        float3 diffuse;
        float3 specular;
        if(metallic >= 0.99f)
        {
            fresnel = Kd_val * FrConductor(dot(prd_radiance.direction, normalize(-ray.direction + prd_radiance.direction)), prd_radiance.current_index_of_refraction, index_of_refraction, absorption_coefficien);
           
            diffuse = make_float3(0.0f);
            
            specular = TorranceSparrow_f(Kn_val, -ray.direction, prd_radiance.direction, fresnel, roughness);
        }
        else if(metallic <= 0.01f)
        {
            fresnel = make_float3(FrDielectric(dot(prd_radiance.direction, normalize(-ray.direction + prd_radiance.direction)), prd_radiance.current_index_of_refraction, index_of_refraction));
            
            diffuse = OrenNayar_full_f(Kd_val, Kn_val, -ray.direction, prd_radiance.direction, roughness);
            diffuse = diffuse * (make_float3(1.0f) - fresnel);
            
            specular = TorranceSparrow_f(Kn_val, -ray.direction, prd_radiance.direction, fresnel, roughness);
        }
        else
        {
            float R0 = (prd_radiance.current_index_of_refraction - index_of_refraction) / (prd_radiance.current_index_of_refraction + index_of_refraction);
            R0 *= R0;
            float3 F0 = lerp(make_float3(R0), Kd_val, metallic);
            fresnel = SchlickFresnel(dot(normalize(-ray.direction + prd_radiance.direction), prd_radiance.direction), F0);

            diffuse = OrenNayar_full_f(Kd_val, Kn_val, -ray.direction, prd_radiance.direction, roughness);
            diffuse = diffuse * (make_float3(1.0f) - fresnel) * (1.0f - metallic);
            
            specular = TorranceSparrow_f(Kn_val, -ray.direction, prd_radiance.direction, fresnel, roughness);
        }

        // http://www.rorydriscoll.com/2009/01/07/better-sampling/
        prd_radiance.attenuation = ((diffuse + specular) * prd_radiance.attenuation) * 2.0f * M_PIf * NdotL;
    }
    else
    {
        prd_radiance.attenuation = make_float3(0.0f);
    }
}

//-----------------------------------------------------------------------------
//
//  Exception program
//
//-----------------------------------------------------------------------------

RT_PROGRAM void exception()
{
	const unsigned int code = rtGetExceptionCode();
	rtPrintf( "Caught exception 0x%X at launch index (%d,%d)\n", code, launch_index.x, launch_index.y );
    output_buffer[launch_index] = make_float4(bad_color, 1.0f);
}

//-----------------------------------------------------------------------------
//
//  Miss program
//
//-----------------------------------------------------------------------------

RT_PROGRAM void miss()
{
    prd_radiance.radiance = prd_radiance.attenuation * bg_color;
    prd_radiance.done = true;
}

//
// Environment map background
//
rtTextureSampler<float4, 2> envmap;
RT_PROGRAM void envmap_miss()
{
    float theta = atan2f( ray.direction.x, ray.direction.z );
    float phi   = M_PIf * 0.5f -  acosf( ray.direction.y );
    float u     = (theta + M_PIf) * (0.5f * M_1_PIf);
    float v     = 0.5f * ( 1.0f + sinf(phi) );
    prd_radiance.radiance = prd_radiance.attenuation * make_float3( tex2D(envmap, u, v) );
    prd_radiance.done = true;
}