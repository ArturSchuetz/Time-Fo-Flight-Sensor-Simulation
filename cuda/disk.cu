#include "hip/hip_runtime.h"
/* 
 * Copyright (c) 2018 NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disklaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disklaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE diskLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include <optix_world.h>
#include "intersection_refinement.h"

using namespace optix;

rtDeclareVariable(float4,  disk_position, , );
rtDeclareVariable(float4,  disk_normal, , );

rtDeclareVariable(float3, texcoord,         attribute texcoord, ); 
rtDeclareVariable(float3, geometric_normal, attribute geometric_normal, ); 
rtDeclareVariable(float3, shading_tangent,  attribute shading_tangent, ); 
rtDeclareVariable(float3, shading_bitangent, attribute shading_bitangent, ); 
rtDeclareVariable(float3, shading_normal,   attribute shading_normal, ); 

rtDeclareVariable(float3, back_hit_point,   attribute back_hit_point, ); 
rtDeclareVariable(float3, front_hit_point,  attribute front_hit_point, ); 

rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );

bool intersectPlane(const float3 &n, const float3 &p0, const float3 &l0, const float3 &l, float &t) 
{ 
  float denom = dot(n, l); 
  if (denom > 1e-6) 
  { 
    float3 p0l0 = p0 - l0; 
    t = dot(p0l0, n) / denom; 
    return (t >= 0.0f); 
  }
  return false; 
} 

 template<bool use_robust_method>
 static __device__
 void intersect_disk(void)
 {
  float3 center = make_float3(disk_position);
  float3 O = ray.origin - center;
  float3 D = ray.direction;
  float radius = disk_position.w;

  float t = 0; 
  if (intersectPlane(n, p0, l0, l, t))
  { 
    float3 p = l0 + l * t; 
    float3 v = p - p0; 
    float d2 = dot(v, v); 
    if (d2 <= radius * radius)
    {

    }
  }
 }
 
 
 RT_PROGRAM void intersect(int primIdx)
 {
  intersect_disk<false>();
 }
 
 
 RT_PROGRAM void robust_intersect(int primIdx)
 {
  intersect_disk<true>();
 }
 
 
 RT_PROGRAM void bounds (int, float result[6])
 {
   const float3 cen = make_float3( sphere );
   const float3 rad = make_float3( sphere.w );
 
   optix::Aabb* aabb = (optix::Aabb*)result;
   
   if( rad.x > 0.0f  && !isinf(rad.x) ) {
     aabb->m_min = cen - rad;
     aabb->m_max = cen + rad;
   } else {
     aabb->invalidate();
   }
 }
 
 