#include "hip/hip_runtime.h"
/* 
* Copyright (c) 2018 NVIDIA CORPORATION. All rights reserved.
*
* Redistribution and use in source and binary forms, with or without
* modification, are permitted provided that the following conditions
* are met:
*  * Redistributions of source code must retain the above copyright
*    notice, this list of conditions and the following disclaimer.
*  * Redistributions in binary form must reproduce the above copyright
*    notice, this list of conditions and the following disclaimer in the
*    documentation and/or other materials provided with the distribution.
*  * Neither the name of NVIDIA CORPORATION nor the names of its
*    contributors may be used to endorse or promote products derived
*    from this software without specific prindex_of_refraction written permission.
*
* THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
* EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
* IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
* PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
* CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
* EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
* PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
* PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
* OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
* (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
* OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
*/

#include <optix.h>
#include <optix_math.h>
#include <optixu/optixu_math_namespace.h>
#include <optixu/optixu_matrix_namespace.h>
#include "random.h"
#include "helpers.h"
#include "microfacet.h"

struct PerRayData_radiance
{
    float3 result;
    float2 ir_result_pulse;
    float4 ir_result_rect;
    float4 ir_result_sin;

    float3 radiance;
    float2 ir_radiance_pulse;
    float4 ir_radiance_rect;
    float4 ir_radiance_sin;

    float3 attenuation;
    float ir_attenuation;
    float ir_traveledDistance;

    float3 origin;
    float3 direction;

    float current_index_of_refraction;
    unsigned int seed;
    int depth;
    int done;
};

struct PerRayData_shadow
{
    bool inShadow;
};

struct BasicLight
{
    #if defined(__cplusplus)
    typedef optix::float3 float3;
    #endif
    float3 pos;
    float3 color;
    float3 direction;
    float   intensity;
    int     casts_shadow; 
};

using namespace optix;

// Scene wide variables

rtDeclareVariable(uint2, launch_index, rtLaunchIndex, );
rtDeclareVariable(uint2, launch_dim,   rtLaunchDim, );
rtDeclareVariable(float, time_view_scale, , ) = 1e-6f;


//-----------------------------------------------------------------------------
//  Camera program -- main ray tracing loop
//-----------------------------------------------------------------------------

rtDeclareVariable(float3,        eye, , );
rtDeclareVariable(float3,        U, , );
rtDeclareVariable(float3,        V, , );
rtDeclareVariable(float3,        W, , );
rtDeclareVariable(float3,        bad_color, , );
rtDeclareVariable(float3, 		 bg_color, , );
rtDeclareVariable(unsigned int,  frame_number, , );
rtDeclareVariable(unsigned int,  sqrt_num_samples, , );
rtDeclareVariable(unsigned int,  rr_begin_depth, , );
rtDeclareVariable(unsigned int,  max_depth, , );

rtBuffer<float4, 2> input_rayDirections;

rtBuffer<float4, 2> output_buffer;
rtBuffer<float2, 2> output_buckets_pulse;
rtBuffer<float4, 2> output_buckets_rect;
rtBuffer<float4, 2> output_buckets_sin;

rtDeclareVariable(float3, back_hit_point,   attribute back_hit_point, ); 
rtDeclareVariable(float3, front_hit_point,  attribute front_hit_point, ); 
rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );

rtDeclareVariable(float, t_hit, rtIntersectionDistance, );
rtDeclareVariable(PerRayData_radiance, prd_radiance, rtPayload, );

rtDeclareVariable(float3, geometric_normal, attribute geometric_normal, ); 
rtDeclareVariable(float3, shading_normal, attribute shading_normal, ); 
rtDeclareVariable(float3, shading_tangent, attribute shading_tangent, ); 
rtDeclareVariable(float3, shading_bitangent, attribute shading_bitangent, ); 

rtBuffer<BasicLight>                 lights;
rtBuffer<BasicLight>                 ir_lights;

rtDeclareVariable(unsigned int,      radiance_ray_type, , );
rtDeclareVariable(unsigned int,      shadow_ray_type, , );
rtDeclareVariable(float,             scene_epsilon, , );
rtDeclareVariable(rtObject,          top_object, , );
rtDeclareVariable(rtObject,          top_shadower, , );

rtDeclareVariable(float,  frequency, , );
const float speedOfLight = 299792458.0f;
const float smallest_value = 0.0001f; 

//-----------------------------------------------------------------------------
//
//  Camera program -- main ray tracing loop
//
//-----------------------------------------------------------------------------

RT_PROGRAM void pathtrace_camera()
{
    size_t2 screen = output_buffer.size();

    float2 inv_screen = 1.0f/make_float2(screen) * 2.f;
    float2 pixel = (make_float2(launch_index)) * inv_screen - 1.f;

    float2 jitter_scale = inv_screen / sqrt_num_samples;
    unsigned int samples_per_pixel = sqrt_num_samples * sqrt_num_samples;
    float3 result = make_float3(0.0f);
    float2 ir_result_pulse = make_float2(0.0f);
    float4 ir_result_rect = make_float4(0.0f);
    float4 ir_result_sin = make_float4(0.0f);

    size_t2 bufferSize = input_rayDirections.size();
    unsigned int seed = tea<16>(screen.x * launch_index.y + launch_index.x, frame_number);

    do 
    {
        //
        // Sample pixel
        //
        float2 jitter = make_float2(0.0f);
        do
        {
            jitter = make_float2(rnd(seed) - 0.5f, rnd(seed) - 0.5f);
        } while(sqrt((jitter.x * jitter.x) + (jitter.y * jitter.y)) > 0.5f);

        float2 d = pixel + (jitter * inv_screen);

        if(d.x > 1.0f) d.x = 1.0f;
        if(d.y > 1.0f) d.y = 1.0f;
        if(d.x < -1.0f) d.x = -1.0f;
        if(d.y < -1.0f) d.y = -1.0f;

        //
        // Sample pixel using jittering
        //
        /*
        unsigned int x = samples_per_pixel % sqrt_num_samples;
        unsigned int y = samples_per_pixel / sqrt_num_samples;
        float2 jitter = make_float2(x - rnd(seed), y - rnd(seed));
        float2 d = pixel + jitter * jitter_scale;
        */

        float2 relativeCoordinate = ((d + 1.0f) / 2.0f);
        float3 calculated_ray = make_float3(input_rayDirections[make_uint2(relativeCoordinate.x * (bufferSize.x - 1), relativeCoordinate.y * (bufferSize.y - 1))]);

        float3 ray_origin = eye;
        float3 ray_direction = normalize(calculated_ray.x*U + calculated_ray.y*V + -calculated_ray.z*W);

        // Initialze per-ray data
        PerRayData_radiance prd;
        prd.result = make_float3(0.f);
        prd.ir_result_pulse = make_float2(0.f);
        prd.ir_result_rect = make_float4(0.f);
        prd.ir_result_sin = make_float4(0.f);

        prd.attenuation = make_float3(1.f);
        prd.ir_attenuation = 1.0f;

        prd.ir_traveledDistance = 0.0f;

        prd.current_index_of_refraction = 1.00029f; // ior of air
        prd.seed = seed;
        prd.depth = 0;
        prd.done = false;

        // Each iteration is a segment of the ray path.  The closest hit will
        // return new segments to be traced here.
        for(;;)
        {
            Ray ray = make_Ray(ray_origin, ray_direction, radiance_ray_type, scene_epsilon, RT_DEFAULT_MAX);
            rtTrace(top_object, ray, prd);

            // Russian roulette termination 
            if(prd.depth >= rr_begin_depth)
            {
                float pcont = fmaxf(prd.attenuation);
                if(rnd(prd.seed) >= pcont)
                    break;
                    
                prd.ir_attenuation /= prd.ir_attenuation;
                prd.attenuation /= pcont;
            }

            prd.depth++;
            prd.result += prd.radiance;
            prd.ir_result_pulse += (prd.ir_radiance_pulse);
            prd.ir_result_rect += (prd.ir_radiance_rect);
            prd.ir_result_sin += (prd.ir_radiance_sin);

            if(prd.done || prd.depth >= max_depth)
            {
                break;
            }

            // Update ray data for the next path segment
            ray_origin = prd.origin;
            ray_direction = prd.direction;
        }

        result += prd.result;
        ir_result_pulse += prd.ir_result_pulse;
        ir_result_rect += prd.ir_result_rect;
        ir_result_sin += prd.ir_result_sin;

        seed = prd.seed;
    } while (--samples_per_pixel);

    //
    // Update the output buffer
    //
    float3 pixel_color      = result/(sqrt_num_samples*sqrt_num_samples);
    float2 ir_buckets_pulse = ir_result_pulse/(sqrt_num_samples*sqrt_num_samples);
    float4 ir_buckets_rect  = ir_result_rect/(sqrt_num_samples*sqrt_num_samples);
    float4 ir_buckets_sin   = ir_result_sin/(sqrt_num_samples*sqrt_num_samples);

    if (frame_number > 1)
    {
        float a = 1.0f / (float)frame_number;
        float3 old_color = make_float3(output_buffer[launch_index]);
        output_buffer[launch_index] = make_float4( lerp( old_color, pixel_color, a ), 1.0f);
        
        float2 old_ir_buckets_pulse = output_buckets_pulse[launch_index];
        output_buckets_pulse[launch_index] = lerp( old_ir_buckets_pulse, ir_buckets_pulse, a );

        float4 old_ir_buckets_rect = output_buckets_rect[launch_index];
        output_buckets_rect[launch_index] = lerp( old_ir_buckets_rect, ir_buckets_rect, a );
        
        float4 old_ir_buckets_sin = output_buckets_sin[launch_index];
        output_buckets_sin[launch_index] = lerp( old_ir_buckets_sin, ir_buckets_sin, a );
    }
    else
    {
        output_buffer[launch_index] = make_float4(pixel_color, 1.0f);
        output_buckets_pulse[launch_index] = ir_buckets_pulse;
        output_buckets_rect[launch_index] = ir_buckets_rect;
        output_buckets_sin[launch_index] = ir_buckets_sin;
    }
}

rtTextureSampler<float4, 2> Kd_map;
rtTextureSampler<float4, 2> Ks_map;
rtTextureSampler<float4, 2> Kn_map;
rtTextureSampler<float4, 2> Tr_map;
rtTextureSampler<float4, 2> Pm_map;
rtTextureSampler<float4, 2> Ke_map;
rtDeclareVariable(float3, texcoord, attribute texcoord, ); 
rtDeclareVariable(float, index_of_refraction, , );
rtDeclareVariable(float, absorption_coefficien, , );

//-----------------------------------------------------------------------------
//
//  Shadow any-hit
//
//-----------------------------------------------------------------------------

rtDeclareVariable(PerRayData_shadow,   prd_shadow, rtPayload, );

RT_PROGRAM void any_hit_shadow()
{
    const float3 Tr_val = make_float3( tex2D( Tr_map, texcoord.x, texcoord.y ) );
    if(Tr_val.x < 1.0f)
    {
        rtIgnoreIntersection();
    }
    else
    {
        prd_shadow.inShadow = true;
        rtTerminateRay();
    }
}

float getArea(float amplitude, float frequency, float offset, float from, float to)
{
	return ((amplitude * (cosf(2.0f * M_PIf * frequency * from) - cosf(2.0f * M_PIf * frequency * to))) / (2.0f * M_PIf * frequency)) - (offset * from) + (offset * to);
}

//-----------------------------------------------------------------------------
//
//  Cook-Sparrow and Oren-Nayar surface closest-hit
//  http://www.pbr-book.org/3ed-2018/Reflection_Models/Microfacet_Models.html
//
//-----------------------------------------------------------------------------

RT_PROGRAM void microfacet_closest_hit()
{
    float3 world_geometric_normal	= normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, geometric_normal ) );
    float3 world_shading_tangent	= normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, shading_tangent ) );
    float3 world_shading_bitangent	= normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, shading_bitangent ) );
    float3 world_shading_normal		= normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, shading_normal ) );

    float3 fftangent = faceforward( world_shading_tangent, -ray.direction, world_geometric_normal );
    float3 ffbitangent = faceforward( world_shading_bitangent, -ray.direction, world_geometric_normal );
    float3 ffnormal = faceforward( world_shading_normal, -ray.direction, world_geometric_normal );

    Matrix<3, 3> tangentMatrix;
    tangentMatrix.setCol(0, fftangent);
    tangentMatrix.setCol(1, ffbitangent);
    tangentMatrix.setCol(2, ffnormal);

    const float3 Kd_val = make_float3( tex2D( Kd_map, texcoord.x, texcoord.y ) );
    const float3 Ks_val = make_float3( tex2D( Ks_map, texcoord.x, texcoord.y ) );
          float3 Kn_val = normalize(make_float3( tex2D( Kn_map, texcoord.x, texcoord.y ) ) * 2.0f - 1.0f);
    const float3 Tr_val = make_float3( tex2D( Tr_map, texcoord.x, texcoord.y ) );
    const float3 Pm_val = make_float3( tex2D( Pm_map, texcoord.x, texcoord.y ) );
    const float3 Ke_val = make_float3( tex2D( Ke_map, texcoord.x, texcoord.y ) );

    Kn_val = tangentMatrix * make_float3(Kn_val.x, Kn_val.y, Kn_val.z);

    if(length(Kn_val) > 0)
        Kn_val = normalize(Kn_val);
    else
        Kn_val = ffnormal;

    if(Tr_val.x < 1.0f)
    {       
        if(prd_radiance.depth > 0 && Tr_val.x == 0.0f)
            prd_radiance.depth = prd_radiance.depth - 1;

        prd_radiance.attenuation = prd_radiance.attenuation * (1.0f - Tr_val.x);
        prd_radiance.ir_attenuation = prd_radiance.ir_attenuation * (1.0f - Tr_val.x);
        const float3 bhp = rtTransformPoint(RT_OBJECT_TO_WORLD, back_hit_point);
        prd_radiance.ir_traveledDistance += length(ray.origin - bhp);

        prd_radiance.origin = bhp;
        prd_radiance.direction = ray.direction;

        prd_radiance.radiance = make_float3(0.0f);
        prd_radiance.ir_radiance_pulse = make_float2(0.0f);
        prd_radiance.ir_radiance_rect = make_float4(0.0f);
        prd_radiance.ir_radiance_sin = make_float4(0.0f);
        return;
    }
    
    float roughness = saturate(1.0f - Ks_val.x);
    float metallic = Pm_val.x;

    float3 hit_point = ray.origin + (t_hit * ray.direction);
    prd_radiance.ir_traveledDistance += length(ray.origin - hit_point);

    //
    // Next event estimation (compute direct lighting).
    //
    float3 result = make_float3(0.0f);
    float2 ir_result_pulse = make_float2(0.0f);
    float4 ir_result_rect = make_float4(0.0f);
    float4 ir_result_sin = make_float4(0.0f);
    
    unsigned int num_lights = lights.size();
    for(int i = 0; i < num_lights; ++i)
    {
        BasicLight light = lights[i];
        float3 lightDir = light.pos - hit_point;
        const float LightdistPow2 = dot(lightDir, lightDir);
        const float Lightdist = sqrt(LightdistPow2);
        lightDir = lightDir / Lightdist;

        const float NdotL = saturate(dot(Kn_val, lightDir));

        // cast shadow ray
        if ( NdotL > smallest_value)
        {
            PerRayData_shadow shadow_prd;
            shadow_prd.inShadow = false;
            Ray shadow_ray = make_Ray( hit_point, lightDir, shadow_ray_type, scene_epsilon, Lightdist - scene_epsilon);
            rtTrace(top_object, shadow_ray, shadow_prd);

            if(!shadow_prd.inShadow)
            {
                float3 fresnel;
                if(metallic >= 0.99f)
                {
                    fresnel = Kd_val * FrConductor(dot(-ray.direction, lightDir), prd_radiance.current_index_of_refraction, index_of_refraction, absorption_coefficien);
                }
                else if(metallic <= 0.01f)
                {
                    fresnel = make_float3(FrDielectric(dot(-ray.direction, lightDir), prd_radiance.current_index_of_refraction, index_of_refraction));
                }
                else
                {
                    fresnel = make_float3(0.0f);
                }
        
                // =====================================
                // Visible light
                float3 diffuse = OrenNayar_full_f(Kd_val, Kn_val, -ray.direction, lightDir, roughness);
                diffuse = diffuse * (make_float3(1.0f) - fresnel) * (1.0f - metallic);

                float3 specular = TorranceSparrow_f(Kn_val, -ray.direction, lightDir, fresnel, roughness);
            
                float3 temp = prd_radiance.attenuation * (diffuse + specular) * (light.color * light.intensity * (NdotL / LightdistPow2));
                result += temp;

                float ir_radiance = luminanceCIE(temp) * 0.5f;
                ir_result_pulse += make_float2(ir_radiance, ir_radiance);
                ir_result_sin += make_float4(ir_radiance, ir_radiance, ir_radiance, ir_radiance);
                ir_radiance = ir_radiance * 0.5f;
                ir_result_rect += make_float4(ir_radiance, ir_radiance, ir_radiance, ir_radiance);
            }
        }
    }
    
    const double pulselength = (1.0f / frequency) * 0.5f;
    
    unsigned int num_ir_lights = ir_lights.size();
    for(int i = 0; i < num_ir_lights; ++i)
    {
        BasicLight light = ir_lights[i];
        float3 lightDir = light.pos - hit_point;
        const float LightdistPow2 = dot(lightDir, lightDir);
        const float Lightdist = sqrt(LightdistPow2);
        lightDir = lightDir / Lightdist;

        const float NdotL = saturate(dot(Kn_val, lightDir));

        if ( NdotL > smallest_value && dot(light.direction, -lightDir) > 0.0f)
        {
            PerRayData_shadow shadow_prd;
            shadow_prd.inShadow = false;
            // Note: bias both ends of the shadow ray, in case the light is also present as geometry in the scene.
            Ray shadow_ray = make_Ray( hit_point, lightDir, shadow_ray_type, scene_epsilon, Lightdist - scene_epsilon );
            rtTrace(top_object, shadow_ray, shadow_prd);

            if(!shadow_prd.inShadow)
            {
                float3 fresnel;
                if(metallic >= 0.99f)
                {
                    fresnel = Kd_val * FrConductor(dot(-ray.direction, lightDir), prd_radiance.current_index_of_refraction, index_of_refraction, absorption_coefficien);
                }
                else if(metallic <= 0.01f)
                {
                    fresnel = make_float3(FrDielectric(dot(-ray.direction, lightDir), prd_radiance.current_index_of_refraction, index_of_refraction));
                }
                else
                {
                    fresnel = make_float3(0.0f);
                }

                float3 ir_diffuse = OrenNayar_full_f(Kd_val, Kn_val, -ray.direction, lightDir, roughness);
                ir_diffuse = ir_diffuse * (make_float3(1.0f) - fresnel) * (1.0f - metallic);
                
                float3 ir_specular = TorranceSparrow_f(Kn_val, -ray.direction, lightDir, fresnel, roughness);
        
                float Intensity = prd_radiance.ir_attenuation * luminanceCIE((ir_diffuse + ir_specular) * (light.color * light.intensity * (NdotL / LightdistPow2)));

                // IR Calculations
                float sourceToSensorDistance = Lightdist + prd_radiance.ir_traveledDistance;
                float deltaTime = sourceToSensorDistance / speedOfLight;

                float C1_start = 0.0f;
                float C2_start = pulselength;

                float C3_start = pulselength * 0.5f;
                float C4_start = (pulselength * 0.5f) + pulselength;
                
                // ==================================================
                // Lichtpuls
                {
                    float begin = deltaTime;
                    float end = deltaTime + pulselength;

                    if((end >= C1_start && end <= C1_start + pulselength) || (begin >= C1_start && begin <= C1_start + pulselength))
                    {
                        if(begin > C1_start)
                            ir_result_pulse.x += (((C1_start + pulselength) - begin) / pulselength) * Intensity;
                        else
                            ir_result_pulse.x += ((end - C1_start) / pulselength) * Intensity;
                    }
                    
                    if((end >= C2_start && end <= C2_start + pulselength) || (begin >= C2_start && begin <= C2_start + pulselength))
                    {
                        if(begin > C2_start)
                            ir_result_pulse.y += (((C2_start + pulselength) - begin) / pulselength) * Intensity;
                        else
                            ir_result_pulse.y += ((end - C2_start) / pulselength) * Intensity;
                    }
                }

                // ==================================================
                // Sinus Welle
                ir_result_sin.x += Intensity * getArea(frequency, frequency, frequency, C1_start - deltaTime, (C1_start - deltaTime) + pulselength);
                ir_result_sin.y += Intensity * getArea(frequency, frequency, frequency, C2_start - deltaTime, (C2_start - deltaTime) + pulselength);
                ir_result_sin.z += Intensity * getArea(frequency, frequency, frequency, C3_start - deltaTime, (C3_start - deltaTime) + pulselength);
                ir_result_sin.w += Intensity * getArea(frequency, frequency, frequency, C4_start - deltaTime, (C4_start - deltaTime) + pulselength);

                // ==================================================
                // Rechteck Welle
                while(deltaTime < C4_start + pulselength)
                {
                    deltaTime = deltaTime + (pulselength * 2.0);
                }
                
                while(deltaTime + pulselength > 0.0f)
                {
                    float begin = deltaTime;
                    float end = deltaTime + pulselength;

                    if((end >= C1_start && end <= C1_start + pulselength) || (begin >= C1_start && begin <= C1_start + pulselength))
                    {
                        if(begin > C1_start)
                            ir_result_rect.x += (((C1_start + pulselength) - begin) / pulselength) * Intensity;
                        else
                            ir_result_rect.x += ((end - C1_start) / pulselength) * Intensity;
                    }
                    
                    if((end >= C2_start && end <= C2_start + pulselength) || (begin >= C2_start && begin <= C2_start + pulselength))
                    {
                        if(begin > C2_start)
                            ir_result_rect.y += (((C2_start + pulselength) - begin) / pulselength) * Intensity;
                        else
                            ir_result_rect.y += ((end - C2_start) / pulselength) * Intensity;
                    }

                    if((end >= C3_start && end <= C3_start + pulselength) || (begin >= C3_start && begin <= C3_start + pulselength))
                    {
                        if(begin > C3_start)
                            ir_result_rect.z += (((C3_start + pulselength) - begin) / pulselength) * Intensity;
                        else
                            ir_result_rect.z += ((end - C3_start) / pulselength) * Intensity;
                    }
                    
                    if((end >= C4_start && end <= C4_start + pulselength) || (begin >= C4_start && begin <= C4_start + pulselength))
                    {
                        if(begin > C4_start)
                            ir_result_rect.w += (((C4_start + pulselength) - begin) / pulselength) * Intensity;
                        else
                            ir_result_rect.w += ((end - C4_start) / pulselength) * Intensity;
                    }

                    deltaTime = deltaTime - (pulselength * 2.0);
                }
            }
        }
    }
    
    prd_radiance.ir_radiance_pulse = ir_result_pulse;
    prd_radiance.ir_radiance_rect = ir_result_rect;
    prd_radiance.ir_radiance_sin = ir_result_sin;
    prd_radiance.radiance = result;

    if(Ke_val.x > 0.0f || Ke_val.y > 0.0f || Ke_val.z > 0.0f)
    {
        float intensity = luminanceCIE(Ke_val) * 0.5f;
        prd_radiance.ir_radiance_pulse += prd_radiance.ir_attenuation * make_float2(intensity, intensity);
        prd_radiance.ir_radiance_sin += prd_radiance.ir_attenuation * make_float4(intensity, intensity, intensity, intensity);
        intensity = intensity * 0.5f;
        prd_radiance.ir_radiance_rect += prd_radiance.ir_attenuation * make_float4(intensity, intensity, intensity, intensity);
        prd_radiance.radiance += prd_radiance.attenuation * Ke_val;
    }
    
    //
    // Generate a reflection ray.  This will be traced back in ray-gen.
    //
    prd_radiance.origin = hit_point;

    float3 p;

    optix::Onb onb( Kn_val );
    float z1=rnd(prd_radiance.seed);
    float z2=rnd(prd_radiance.seed);
    //random_sample_hemisphere(z1, z2, p);
    cosine_sample_hemisphere(z1, z2, p);

    onb.inverse_transform(p);    
    prd_radiance.direction = normalize(p);
        
    const float NdotL = saturate(dot(Kn_val, prd_radiance.direction));
    if (NdotL > smallest_value)
    {
        float3 fresnel;
        float3 diffuse;
        float3 specular;
        if(metallic >= 0.99f)
        {
            fresnel = Kd_val * FrConductor(dot(prd_radiance.direction, normalize(-ray.direction + prd_radiance.direction)), prd_radiance.current_index_of_refraction, index_of_refraction, absorption_coefficien);
           
            diffuse = make_float3(0.0f);
            
            specular = TorranceSparrow_f(Kn_val, -ray.direction, prd_radiance.direction, fresnel, roughness);
        }
        else if(metallic <= 0.01f)
        {
            fresnel = make_float3(FrDielectric(dot(prd_radiance.direction, normalize(-ray.direction + prd_radiance.direction)), prd_radiance.current_index_of_refraction, index_of_refraction));
            
            diffuse = OrenNayar_full_f(Kd_val, Kn_val, -ray.direction, prd_radiance.direction, roughness);
            diffuse = diffuse * (make_float3(1.0f) - fresnel);
            
            specular = TorranceSparrow_f(Kn_val, -ray.direction, prd_radiance.direction, fresnel, roughness);
        }
        else
        {
            float R0 = (prd_radiance.current_index_of_refraction - index_of_refraction) / (prd_radiance.current_index_of_refraction + index_of_refraction);
            R0 *= R0;
            float3 F0 = lerp(make_float3(R0), Kd_val, metallic);
            fresnel = SchlickFresnel(dot(normalize(-ray.direction + prd_radiance.direction), prd_radiance.direction), F0);
            
            diffuse = OrenNayar_full_f(Kd_val, Kn_val, -ray.direction, prd_radiance.direction, roughness);
            diffuse = diffuse * (make_float3(1.0f) - fresnel) * (1.0f - metallic);
            
            specular = TorranceSparrow_f(Kn_val, -ray.direction, prd_radiance.direction, fresnel, roughness);
        }
        
        prd_radiance.attenuation = (diffuse + specular) * prd_radiance.attenuation * M_PIf;

        float3 ir_specular = specular;
        prd_radiance.ir_attenuation = luminanceCIE(diffuse + ir_specular) * prd_radiance.ir_attenuation * M_PIf;
    }
    else
    {
        prd_radiance.attenuation = make_float3(0.0f);
        prd_radiance.ir_attenuation = 0.0f;
    }
}

//-----------------------------------------------------------------------------
//
//  Exception program
//
//-----------------------------------------------------------------------------

RT_PROGRAM void exception()
{
	const unsigned int code = rtGetExceptionCode();
	rtPrintf( "Caught exception 0x%X at launch index (%d,%d)\n", code, launch_index.x, launch_index.y );
    output_buffer[launch_index] = make_float4(bad_color, 1.0f);
}

//-----------------------------------------------------------------------------
//
//  Miss program
//
//-----------------------------------------------------------------------------

RT_PROGRAM void miss()
{
    float intensity = luminanceCIE(bg_color) * 0.5f;
    prd_radiance.ir_radiance_pulse = prd_radiance.ir_attenuation * make_float2(intensity, intensity);
    prd_radiance.ir_radiance_sin = prd_radiance.ir_attenuation * make_float4(intensity, intensity, intensity, intensity);
    intensity = intensity * 0.5f;
    prd_radiance.ir_radiance_rect = prd_radiance.ir_attenuation * make_float4(intensity, intensity, intensity, intensity);

    prd_radiance.radiance = prd_radiance.attenuation * bg_color;
    prd_radiance.done = true;
}

//
// Environment map background
//
rtTextureSampler<float4, 2> envmap;
RT_PROGRAM void envmap_miss()
{
    float theta = atan2f( ray.direction.x, ray.direction.z );
    float phi   = M_PIf * 0.5f -  acosf( ray.direction.y );
    float u     = (theta + M_PIf) * (0.5f * M_1_PIf);
    float v     = 0.5f * ( 1.0f + sinf(phi) );

    float intensity = luminanceCIE(make_float3( tex2D(envmap, u, v) )) * 0.5f;
    prd_radiance.ir_radiance_pulse = prd_radiance.ir_attenuation * make_float2(intensity, intensity);
    prd_radiance.ir_radiance_sin = prd_radiance.ir_attenuation * make_float4(intensity, intensity, intensity, intensity);
    intensity = intensity * 0.5f;
    prd_radiance.ir_radiance_rect = prd_radiance.ir_attenuation * make_float4(intensity, intensity, intensity, intensity);

    prd_radiance.radiance = prd_radiance.attenuation * make_float3( tex2D(envmap, u, v) );
    prd_radiance.done = true;
}
